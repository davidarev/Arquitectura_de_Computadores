#include "hip/hip_runtime.h"
/*
 * ¡¡¡ACLARACIÓN A TENER EN CUENTA!!!
 * El código NO ha sido probado en un equipo con una tarjeta gráfica NVIDIA,
 * por lo que no se puede asegurar su correcto funcionamiento.
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define N 10000000

__global__ void kernel_suma(float *arr1, float *arr2, int dim){ // Función kernel que suma dos vectores de dimensión dim y guarda el resultado en arr1
    int id = threadIdx.x + (blockIdx.x * blockDim.x); // id es igual al número de hilo que se está ejecutando en el momento de la llamada a la función
    if (id < dim) arr1[id] = arr1[id] + arr2[id];
}

int main(){
    float *arr1 = (float *)malloc(N * sizeof(float)); // Reserva de memoria para el array arr1
    float *arr2 = (float *)malloc(N * sizeof(float)); // Reserva de memoria para el array arr2
    float *out = (float *)malloc(N * sizeof(float)); // Reserva de memoria para el array resolver
    float *p_arr1, *p_arr2, *p_out; // Declaración de punteros para los arrays arr1, arr2 y resolver en el dispositivo

    hipMalloc((void **)&p_arr1, sizeof(float) * N); // Reserva de memoria en el dispositivo para el array arr1
    hipMalloc((void **)&p_arr2, sizeof(float) * N); // Reserva de memoria en el dispositivo para el array arr2
    hipMalloc((void **)&p_out, sizeof(float) * N); // Reserva de memoria en el dispositivo para el array resolver

    memset(arr1, 0, sizeof(float) * N); // Inicialización del array arr1.
    memset(arr2, 0, sizeof(float) * N); // Inicialización del array arr2
    for (int i = 0; i < N; i++){
        arr1[i] = 1.0f; // Asignación de valores al array arr1
        arr2[i] = 2.0f; // Asignación de valores al array arr2
    }

    printf("Array A: ");
    for (int i = 0; i < N; i++){
        printf("%d,", arr1[i]); // Impresión del array arr1
    }

    printf("\nArray B: ");
    for (int i = 0; i < N; i++){
        printf("%d,", arr2[i]); // Impresión del array arr2
    }

    hipMemcpy(p_arr1, arr1, sizeof(float) * N, hipMemcpyHostToDevice); // Copia de los datos del array arr1 al array p_arr1 en el dispositivo
    hipMemcpy(p_arr2, arr2, sizeof(float) * N, hipMemcpyHostToDevice); // Copia de los datos del array arr2 al array p_arr2 en el dispositivo
    dim3 nthreads(256); // Variable de tipo dim3 que contiene el número de hilos por bloque, en este caso 256
    dim3 nblocks((N / nthreads.x) + (N % nthreads.x ? 1 : 0)); // Variable de tipo dim3 que contiene el número de bloques, en este caso N/256 + 1
    kernel_suma <<< nblocks.x, nthreads.x >>> (arr1, arr2, N); // Llamada a la función kernel_suma
    hipDeviceSynchronize(); // Espera a que todos los hilos terminen su ejecución
    hipMemcpy(resolver, arr1, sizeof(float) * N, hipMemcpyDeviceToHost); // Copia el resultado de la suma de los arrays arr1 y arr2 al array resolver

    printf("\nResolver: ");
    for (int i = 0; i < N; i++){
        printf("%d,", out[i]); // Impresión del array resolver
    }

    // Liberación de memoria
    free(arr1);
    free(arr2);
    free(out);
    hipFree(p_arr1);
    hipFree(p_arr2);
    hipFree(p_out);

    return 0;
}
